// C standard library
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA APIs
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <sys/time.h>
#define MILLION 1000000.0

// Global constants
#define PI_ACCURATE 3.141592653589794

// Using GPU with id 0
#define GPU_ID 0 
// Number of experiments = 2^23
#define N 8388608 
// Number of threads in a thread block
#define THREADS 2048
// Number of thread blocks in the grid
#define THREAD_BLOCKS (N/THREADS)
// Seed of random number generator
#define RAND_SEED 123456ULL

__global__ void experiment(int *counter, float *x_arr, float *y_arr)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Generate coordinates x, y
    float x = x_arr[index], y = y_arr[index];
    
    printf("index %d, x: %f, y: %f\n", index, x, y);

    if (x * x + y * y <= 1.0f)
    {
        atomicAdd(counter, 1);
        printf("Counter: %d\n", *counter);
    }
}

double wall_clock_time(void)
{


  double secs;
  struct timeval tp;

  gettimeofday(&tp, NULL);
  secs = (MILLION * (double)tp.tv_sec + (double)tp.tv_usec) / MILLION;
  return secs;
}

int main()
{
    // Step 1: Initialise GPU

    // Get the number of GPUs available
    int devCount;
    hipGetDeviceCount(&devCount);

    // Check if we have enough GPUs
    if(devCount <= GPU_ID)
    {
        printf("[ERROR] Cannot initialise GPU %d.\n", GPU_ID);
        return 1;
    }

    // Tell CUDA that we want to use GPU 0
    hipSetDevice(GPU_ID);

    // Step 2: Initialise variables on GPU memory directly
    // No need to transfer data form CPU.
    int *d_area;
    float *d_x, *d_y;
    if (hipMalloc((void **)&d_x, N * sizeof(float)) != hipSuccess ||
        hipMalloc((void **)&d_y, N * sizeof(float)) != hipSuccess ||
        hipMalloc((void **)&d_area, sizeof(int)) != hipSuccess)
    {
        printf("Failed to allocate GPU memory!\n");
        return 2;
    }


    // Step 3: Calculate PI

    double time_start = wall_clock_time();
    
    // Initialise random number generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, RAND_SEED);
    
    // Generate random numbers
    hiprandGenerateUniform(gen, d_x, N);
    hiprandGenerateUniform(gen, d_y, N);
    
    // Launch kernel
    printf("Launch\n");
    experiment<<<THREAD_BLOCKS, THREADS>>>(d_area, d_x, d_y);

    double time_end = wall_clock_time();

    // Step 4: Print results
    int *h_area = (int *)malloc(sizeof(int));
    hipMemcpy(h_area, d_area, sizeof(int), hipMemcpyDeviceToHost);

    float pi = (4.0 * (*h_area)) / (float)N;
    printf("\nPi:\t%f\n", pi);

    float error = fabs(pi - PI_ACCURATE) / PI_ACCURATE * 100;
    printf("Error = %.2f%%\n", error);
    //printf("Time elapsed: %.2f%%\n", error);
    
    return 0;
}
