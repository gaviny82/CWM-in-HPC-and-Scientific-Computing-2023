// In this assignment you will write a basic kernel where every thread 
// will write out to console string "Hello world!".
// You will also initialize GPU using cudaSetDevice() and also launch
// your "Hello world" kernel.

#include <stdio.h>
#include <stdlib.h>
// we have to include few more things
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 2: Write a "Hello world" kernel
// 
// Remember that kernel is defined by __global__ and inside it looks like
// a serial code for CPU. For printing out to console you can use printf().

// write your kernel here

//----------------------------------------------------------------------
__global__ void hello_world_GPU(void) {
  printf("Hello world\n");
}

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Initiate GPU using cudaSetDevice()
  //
  // You can also try to write a check if there is a device with that id,
  // so the code behaves nicely when it fails
  
  // write you GPU initialization here

  //----------------------------------------------------------------------

  // Initiate GPU
  int deviceid = 0; // Using GPU with id 0

  // Get the number of GPUs available
  int devCount;
  hipGetDeviceCount(&devCount);

  // Check if we have enough GPUs
  if (devCount <= deviceid)
    return 1;

  // Tell CUDA that we want to use GPU 0
  hipSetDevice(deviceid);

  //----------------------------------------------------------------------
  // TASK 3: execute your "Hello world" kernel on 1 block with 5 threads 
  //         using execution configuration syntax.
  // 
  // You may use whatever syntax version you prefer, a simplified one 
  // dimensional or full three dimensional call using dim3 data type.
  
  // execute your "Hello world" kernel here
  
  //----------------------------------------------------------------------
  hello_world_GPU<<<1, 5>>>();

  hipDeviceReset();
  return (0);
}
